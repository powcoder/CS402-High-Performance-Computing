
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

#define BLOCK_SIZE 16

__global__ 
void mat_mult(float *A, float *B, float *C, int N){
    int row = 0;//TODO
    int col = 0;//TODO

    float sum = 0.0f;
    for (int n = 0; n < N; ++n) {
        sum += A[row*N+n]*B[n*N+col];
    }

    C[row*N+col] = sum;
}

void mat_mult_cpu(float *A, float *B, float *C, int N) {
    #pragma omp parallel for
    for (int row=0; row<N; ++row) {
        for (int col=0; col<N; ++col) {
            float sum = 0.0f;
            for (int n = 0; n < N; ++n){
                sum += A[row*N+n]*B[n*N+col];
            }
            C[row*N+col] = sum;
        }
    }
}

int main(int argc, char *argv[]){
    
    int N,K;
    K = 100;            
    N = K*BLOCK_SIZE;
    
    float *hA,*hB,*hC;
    hA = new float[N*N];
    hB = new float[N*N];
    hC = new float[N*N];

    for (int j=0; j<N; j++){
        for (int i=0; i<N; i++){
            hA[j*N+i] = 2.f*(j+i);
            hB[j*N+i] = 1.f*(j-i);
        }
    }

    int size = N*N*sizeof(float);
    float *dA,*dB,*dC;
    hipMalloc(&dA,size);
    hipMalloc(&dB,size);
    hipMalloc(&dC,size);

    dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 grid(K,K);

    hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
    hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
    
    mat_mult<<<grid,threadBlock>>>(dA,dB,dC,N);
    if (hipPeekAtLastError() != hipSuccess) {
        fprintf(stderr, "CUDA error detected: \"%s\"\n", hipGetErrorString(hipGetLastError()));
        return 1;
    }
    
    float *C;
    C = new float[N*N];
    hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);

    mat_mult_cpu(hA, hB, hC, N);

    for (int row=0; row<N; row++) {
        for (int col=0; col<N; col++) {
            if ( C[row*N+col] != hC[row*N+col] ){
                fprintf(stderr,"Validation failed at row=%d, col=%d.\n", row, col);
                return 1;
            }
        }
    }
}
